// ***********************************************************************
//
// Demo program pro vyuku predmetu APPS (10/2012)
// Petr Olivka, katedra informatiky, FEI, VSB-TU Ostrava
// email:petr.olivka@vsb.cz
//
// Priklad pouziti CUDA technologie.
// Ukazka organizace vlaken v blocich. 
// Kazde vlakno vypise informaci o sve pozici v bloku, 
// a polohu bloku v gridu.
//
// ***********************************************************************

#include <hip/hip_runtime.h>

#include <stdio.h>

// Demo kernel pro zobrazeni hierarchie vlaken
// POZOR! Funkce printf je dostupna od verze compute capability 2.x
__global__ void thread_hierarchy()
{
    // globalni promenne identifikujici vlakno
    // rozmery gridu -			gridDim
	// pozice bloku v gridu -	blockIdx
	// rozmery bloku -			blockDim
	// pozice vlakna v bloku -	threadIdx
    printf( "Block{%d,%d}[%d,%d] Thread{%d,%d}[%d,%d]\n",
	    gridDim.x, gridDim.y, blockIdx.x, blockIdx.y,
		blockDim.x, blockDim.y, threadIdx.x, threadIdx.y );
}

void run_cuda()
{
	hipError_t cerr;
    // nasledujicim prikazem je mozno zvetsit vnitrni buffer pro printf
	/*cerr = cudaDeviceSetLimit( cudaLimitPrintfFifoSize, required_size );
	if ( err != cudaSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, cudaGetErrorString( cerr ) );
    */

	// vytvoreni vlaken se zvolenym kernelem
	// prvni parametr dim3 urcuje rozmer gridu
	// druhy parametr dim3 urcuje rozmer bloku
    thread_hierarchy<<< dim3( 2, 2 ), dim3( 3, 3 )>>>();

	if ( ( cerr = hipGetLastError() ) != hipSuccess )
		printf( "CUDA Error [%d] - '%s'\n", __LINE__, hipGetErrorString( cerr ) );

	// vystupy funkce printf jsou ulozeny v pameti graficke karty,
	// nutno provest synchronizeci, aby se zobrazily
	hipDeviceSynchronize();
}